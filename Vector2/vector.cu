#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <string.h>
#include <unistd.h>
#include "vector.h"

#define checkCudaError(status) { \
	if(status != hipSuccess) { \
		std::cout << "CUDA Error " << __FILE__ << ", " << __LINE__ \
			<< ": " << hipGetErrorString(status) << "\n"; \
		exit(-1); \
	} \
}

__global__ void vecAdd(int * a, int * b, int * c, int size, int k) {

	//ADD CODE HERE
	int i = threadIdx.x;
	int j = blockIdx.x*blockDim.x*k;
	for(int count =0 ; count < k; count++){
		c[i*k + j + count] = a[i*k + j +count] + b[i*k + j + count];
	}
}

void print_help_and_exit(void) {
    printf("Vector CUDA ASS [OPTIONS]\n");
    printf("  -k K\t\tAdditions per thread.\n");
    printf("  -t B\t\tThreads per block less than or equal to 1024.\n");
    printf("  -b S\t\tNo. of blocks.\n");
    printf("  -v V\t\tVector size.\n");
    exit(0);
}

int main(int argc, char *argv[]) {

	int opt;
    uint64_t k = DEFAULT_K;
    uint64_t t = DEFAULT_T;
    uint64_t bl =DEFAULT_B;
    uint64_t v = DEFAULT_V;

    /* Read arguments */ 
    while(-1 != (opt = getopt(argc, argv, "k:t:b:v:h"))) {
        switch(opt) {
        case 'k':
            k = atoi(optarg);
            break;
        case 't':
            t = atoi(optarg);
            break;
        case 'b':
            bl = atoi(optarg);
            break;
        case 'v':
            v = atoi(optarg);
            break;
        case 'h':
            /* Fall through */
        default:
            print_help_and_exit();
            break;
        }
    }

    

	//checkCudaError(hipSetDevice(1));
	int device;
	checkCudaError(hipGetDevice(&device));
	hipDeviceProp_t prop;
	checkCudaError(hipGetDeviceProperties(&prop, device));
	//std::cout << "Device " << device << ": " << prop.name << "\n";
	//std::cout << "GPU/SM Cores: " << prop.multiProcessorCount << "\n";
	//std::cout << "Warp Size: " << prop.warpSize << "\n";
	//std::cout << "Max threads per block: " << prop.maxThreadsPerBlock << "\n";
	//std::cout << "Max ThreadsDim, x: " << prop.maxThreadsDim[0] << ", y: " <<  prop.maxThreadsDim[1] << ", z: " <<  prop.maxThreadsDim[2] << "\n";	
	//std::cout << "Max GridSize, x: " << prop.maxGridSize[0] << ", y: " <<  prop.maxGridSize[1] << ", z: " <<  prop.maxGridSize[2] << "\n";	
	//std::cout << "Total Global Memory: " << (prop.totalGlobalMem>>30) << "TB" << "\n";
	//std::cout << "Shared Memory per Block: " << (prop.sharedMemPerBlock>>10) << "\n";
	//std::cout << "Compute Capability: " << prop.major << "." << prop.minor << "\n";
	

	const uint64_t CTA_SIZE = t;
	const uint64_t THREAD_ADDITIONS = k;
	const uint64_t size = v;
	uint64_t TEMP_GRID_SIZE = size/CTA_SIZE/THREAD_ADDITIONS;
	if(TEMP_GRID_SIZE != bl){
		bl = TEMP_GRID_SIZE;
	}
	const uint64_t GRID_SIZE = bl;
	printf("Vector Add Settings\n");
	std::cout << "Threads per Block\t: " << CTA_SIZE << "\n";
	std::cout << "Thread Block Num\t: " << GRID_SIZE << "\n";
	std::cout << "Additions per block\t: " << THREAD_ADDITIONS << "\n";
	std::cout << "Vector Size\t\t: " << size << "\n";
	
	
	int * a, * b, * c;
	int * dev_a, * dev_b, * dev_c;

	a = (int *) malloc (sizeof(int) * size);
	b = (int *) malloc (sizeof(int) * size);
	c = (int *) malloc (sizeof(int) * size);
	if(!a || !b || !c) {
		std::cout << "Error: out of memory\n";
		exit(-1);
	}

	for(int i = 0; i < size; i++) {
		a[i] = i;
		b[i] = i+1;
	}
	memset(c, 0, sizeof(int) * size);

	checkCudaError(hipMalloc(&dev_a, sizeof(int) * size));
	checkCudaError(hipMalloc(&dev_b, sizeof(int) * size));	
	checkCudaError(hipMalloc(&dev_c, sizeof(int) * size));	
	
	checkCudaError(hipMemcpy(dev_a, a, sizeof(int) * size, hipMemcpyHostToDevice));
	checkCudaError(hipMemcpy(dev_b, b, sizeof(int) * size, hipMemcpyHostToDevice));
	checkCudaError(hipMemset(dev_c, 0, sizeof(int) * size));
	
	hipEvent_t startEvent, stopEvent;
	float elapsedTime;
	
	
	hipEventCreate(&startEvent,0 );
	hipEventCreate(&stopEvent, 0);
	hipEventRecord(startEvent,0);
	
	vecAdd<<<GRID_SIZE, CTA_SIZE>>>(dev_a, dev_b, dev_c, size,THREAD_ADDITIONS);
	hipEventRecord(stopEvent,0);
	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
	checkCudaError(hipDeviceSynchronize());
	
	checkCudaError(hipMemcpy(c, dev_c, sizeof(int) * size, hipMemcpyDeviceToHost));

	for(int i = 0; i < size; i++) {
//		std::cout << i << ": " << c[i] << "\n";
		if(c[i] != i*2+1) {
			std::cout << "Error: c[" << i << "] != " <<
				i*2+1 << "but is: "<< c[i] <<"\n";
			exit(-1);
		}
	}
	std::cout << "Status\t\t\t: Pass\n";
	std::cout << "----- Elapsed Time: " << elapsedTime << " -----" << "\n";
}